#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"
#include <assert.h>
#include <hipblas.h>
#include <hiprand.h>
#include <memory>
#include <iostream>

// kernals
namespace CharacterRecognition {
	using Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}

	void printCuda(float *a1, int n, string name) {
		float *print_a = new float[n];
		cout << name.c_str() << endl;
		cout << "{" << endl;
		hipMemcpy(print_a, a1, n * sizeof(float), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++) {
			cout << "\t" << print_a[i] << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}

	__global__ void bias_addition(int n, float *A, float *B, float *C) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n)
			return;
		C[index] = A[index] + B[index];
	}

	__global__ void relu_activation(int n, float *A, float *C) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n)
			return;
		C[index] = max(0.0f, A[index]);
	}

	__global__ void softmax_activation(int n, float *A, float *C, float exp_sum) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n)
			return;
		C[index] = expf(A[index]) / exp_sum;
	}


	__global__ void scan(int n, float *data, int d) {// function to get sum (for softmax layer)
		int tmp_d = 1 << (d + 1);
		int index = (blockDim.x * blockIdx.x + threadIdx.x)*tmp_d;
		if (index >= n)
			return;
		data[index + tmp_d - 1] += data[index + (tmp_d >> 1) - 1];
	}

	__global__ void exp_copy(int n, float *odata, float *idata) {// kernal to copy exp(idata[i]) to odata[i]
		int index = (blockDim.x * blockIdx.x + threadIdx.x);
		if (index >= n)
			return;
		odata[index] = expf(idata[index]);
	}

	__global__ void gpu_matrix_mult(float *a, float *b, float *c, int m, int n, int k)
	{
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int sum = 0;
		if (col < k && row < m)
		{
			for (int i = 0; i < n; i++)
			{
				sum += a[row * n + i] * b[i * k + col];
			}
			c[row * k + col] = sum;
		}
	}
	__global__ void fill_data(int n, float *data, float val) {
		int index = (blockDim.x * blockIdx.x + threadIdx.x);
		if (index >= n)
			return;
		data[index] = val;
	}

	void Net::GPU_fill_rand(float *A, int size, float std) {
		// Create a pseudo-random number generator
		hiprandGenerator_t prng;
		hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

		// Set the seed for the random number generator using the system clock
		hiprandSetPseudoRandomGeneratorSeed(prng, clock());

		// Fill the array with random numbers on the device
		hiprandGenerateNormal(prng, A, size, 0, std);
	}


	Net::Net(int n, vector<int> layers) : input_size(n), layer_sizes(layers) {
		// layers = {98, 52, 52}
		layers.insert(layers.begin(), n);
		float *dev_w, *dev_b, *dev_g, *dev_a;
		int blocks;
		for (int i = 0; i < layers.size() - 1; i++) {
			hipMalloc((void**)&dev_w, (layers[i] * layers[i + 1]) * sizeof(float));
			hipMalloc((void**)&dev_b, (layers[i + 1]) * sizeof(float));
			// initilize w, b using gaussian distribution
			GPU_fill_rand(dev_w, layers[i] * layers[i + 1], 2.0 / layers[i]); // henormal initilization
			printCuda(dev_w, layers[i] * layers[i + 1], "W fresh");
			GPU_fill_rand(dev_b, layers[i + 1], 0.1f); // zero initilizaton is fine for biases
			// push into vector
			w.push_back(dev_w);
			b.push_back(dev_b);
			// int results
			hipMalloc((void**)&dev_g, (layers[i + 1]) * sizeof(float));
			hipMalloc((void**)&dev_a, (layers[i + 1]) * sizeof(float));
			g.push_back(dev_g);
			a.push_back(dev_a);
		}
		// initilizaton cublas handle
		hipblasCreate(&handle);
	}

	// C(m,n) = A(m,k) * B(k,n)
	void Net::gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
		int lda = m, ldb = k, ldc = m;
		const float alf = 1;
		const float bet = 0;
		const float *alpha = &alf;
		const float *beta = &bet;
		// Do the actual multiplication
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	}

	

	float* Net::forward(float *data, int n) {
		float *res = new float[classes]();
		assert(n == input_size);
		float *dev_data;
		hipMalloc((void**)&dev_data, n * sizeof(float));
		hipMemcpy(dev_data, data, n * sizeof(float), hipMemcpyHostToDevice);
		for (int i = 0; i < layer_sizes.size(); i++) {
			blocks = ceil((layer_sizes[i] + block_size - 1) / block_size);
			// clear g, a for this layer
			hipMemset(g[i], 0, layer_sizes[i] * sizeof(float));
			checkCUDAErrorWithLine("Cuda memset failed!");
			hipMemset(a[i], 0, layer_sizes[i] * sizeof(float));
			checkCUDAErrorWithLine("Cuda memset failed!");
			int block_size_tmp = 16;
			int col_blocks = (1 + block_size_tmp - 1) / block_size_tmp, row_col = (layer_sizes[i]+ block_size_tmp - 1)/ block_size_tmp;
			// matrix multiplication
			if (!i) { // first iteration, so a[i] hasn't been set yet
				gpu_blas_mmul(dev_data, w[i], g[i], 1, input_size, layer_sizes[i]);
				checkCUDAErrorWithLine("gpu mult failed!");
			}
			else {
				gpu_blas_mmul(a[i - 1], w[i], g[i], 1, layer_sizes[i-1], layer_sizes[i]);
				checkCUDAErrorWithLine("gpu mult failed!");
			}
			// bias addition
			bias_addition << <blocks, block_size >> > (layer_sizes[i], g[i], b[i], g[i]); // put result back into g[i]
			checkCUDAErrorWithLine("bias addition failed!");
			if (i != layer_sizes.size() - 1) {
				// relu activation
				relu_activation << <blocks, block_size >> > (layer_sizes[i], g[i], a[i]);
				checkCUDAErrorWithLine("relu failed!");
			}
			else {
				exp_copy << <blocks, block_size >> > (layer_sizes[i], a[i], g[i]);
				checkCUDAErrorWithLine("exp copy failed!");
				// todo optimize this later
				float *tmp = new float[layer_sizes[i]];
				float exp_sum = 0;
				hipMemcpy(tmp, a[i], layer_sizes[i] * sizeof(float), hipMemcpyDeviceToHost);
				for (int pos = 0; pos < layer_sizes[i]; pos++)
					exp_sum += tmp[pos];
				delete[] tmp;
				// modified scan to get the exponential sum of all elements (P1 of assignment used!!)
				//int closest_pow2 = 1 << ilog2ceil(layer_sizes[i]);
				//int blocks_scan;
				//for (int d = 0; d <= ilog2ceil(closest_pow2) - 1; d++) {
				//	// compute number of threads to spawn
				//	blocks_scan = ceil((closest_pow2 / (1 << (d + 1)) + block_size - 1) / block_size);
				//	scan<<<blocks_scan, block_size>>>(closest_pow2, a[i], d);
				//	checkCUDAErrorWithLine("scan failed!");
				//}
				//hipMemcpy(&exp_sum, a[i] + closest_pow2 - 1, sizeof(float), hipMemcpyDeviceToHost);
				// softmax activation
				checkCUDAErrorWithLine("Cuda memcpy failed!");
				softmax_activation << <blocks, block_size >> > (layer_sizes[i], g[i], a[i], exp_sum);
				checkCUDAErrorWithLine("softmax failed!");
			}
		}
		hipMemcpy(res, a[layer_sizes.size() - 1], classes * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorWithLine("Cuda res memcpy failed!");
		return res;
	}


	Net::~Net() {
		// free weights and biases
		for (auto x : w)
			hipFree(x);
		for (auto x : b)
			hipFree(x);
		// intermediate values
		for (auto x : g)
			hipFree(x);
		for (auto x : a)
			hipFree(x);
		// clean culbas hand
		hipblasDestroy(handle);
	}
}
