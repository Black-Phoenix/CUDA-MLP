#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"
#include <assert.h>
#include <hipblas.h>
#include <hiprand.h>
#include <memory>
#include <iostream>

// kernals
namespace CharacterRecognition {
	using Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}

	void printCuda(double *a1, int n, string name) {
		double *print_a = new double[n];
		cout << name.c_str() << endl;
		cout << "{" << endl;
		hipMemcpy(print_a, a1, n * sizeof(double), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++) {
			cout << "\t" << print_a[i] << endl;
		}
		cout << "}" << endl;
		delete[]print_a;
	}
	//////////////////////////////
	/*			KERNALS			*/	
	//////////////////////////////
	__global__ void bias_addition(int n, double *A, double *B, double *C, int sign = 1) { // change sign for subtraction or scaled addition
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n)
			return;
		C[index] = A[index] + sign*B[index];
	}

	__global__ void relu_activation(int n, double *A, double *C) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n)
			return;
		C[index] = max(0.0f, A[index]);
	}
	
	__global__ void relu_grad(int n, double *g, double * grad) { // assumes grad is a 2d array of size n x n
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n)
			return;
		grad[index * n + index] = max(0.0f, g[index]); // makes a diagona matrix
	}

	__global__ void softmax_activation(int n, double *A, double *C, double exp_sum) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n)
			return;
		C[index] = exp(A[index]) / exp_sum;
	}

	__global__ void softmax_grad(int n, double *g, double * grad) {
		int index_i = threadIdx.x + (blockIdx.x * blockDim.x);
		int index_j = threadIdx.y + (blockIdx.y * blockDim.y);
		if (index_i >= n || index_j >= n)
			return;
		grad[index_i * n + index_j] = g[index_i] * ((index_i == index_j) - g[index_j]);
	}

	__global__ void scan(int n, double *data, int d) {// function to get sum (for softmax layer)
		int tmp_d = 1 << (d + 1);
		int index = (blockDim.x * blockIdx.x + threadIdx.x)*tmp_d;
		if (index >= n)
			return;
		data[index + tmp_d - 1] += data[index + (tmp_d >> 1) - 1];
	}

	__global__ void exp_copy(int n, double *odata, double *idata) {// kernal to copy exp(idata[i]) to odata[i]
		int index = (blockDim.x * blockIdx.x + threadIdx.x);
		if (index >= n)
			return;
		odata[index] = exp(idata[index]);
	}

	__global__ void fill_data(int n, double *data, double val) {
		int index = (blockDim.x * blockIdx.x + threadIdx.x);
		if (index >= n)
			return;
		data[index] = val;
	}

	__global__ void gpu_matrix_mult(const double *a, const double *b, double *c, const int m, const int n, const int k)
	{
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int sum = 0; 
		if (col < k && row < m)
		{
			for (int i = 0; i < n; i++)
			{
				sum += a[row * n + i] * b[i * k + col];
			}
			c[row * k + col] = sum;
		}
	}

	//////////////////////////////
	/*			Helper			*/
	//////////////////////////////
	
	void Net::GPU_fill_rand(double *A, int size, double std) {
		// Create a pseudo-random number generator
		hiprandGenerator_t prng;
		hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

		// Set the seed for the random number generator using the system clock
		hiprandSetPseudoRandomGeneratorSeed(prng, clock());

		// Fill the array with random numbers on the device
		hiprandGenerateNormalDouble(prng, A, size, 0, std);
	}

	Net::Net(int n, vector<int> layers) : input_size(n), layer_sizes(layers) {
		// layers = {98, 52, 52}
		// init raw data holder
		hipMalloc((void**)&dev_data, n * sizeof(double));
		layer_count = layers.size();
		layers.insert(layers.begin(), n);
		double *dev_w, *dev_b, *dev_g, *dev_a, *dev_jac;
		int blocks;
		for (int i = 0; i < layer_count; i++) {
			hipMalloc((void**)&dev_w, (layers[i] * layers[i + 1]) * sizeof(double));
			checkCUDAErrorWithLine("Cuda malloc failed!");
			hipMalloc((void**)&dev_b, (layers[i + 1]) * sizeof(double));
			checkCUDAErrorWithLine("Cuda malloc failed!");
			// initilize w, b using gaussian distribution
			GPU_fill_rand(dev_w, layers[i] * layers[i + 1], 2.0 / layers[i]); // uniform random initilization
			//printCuda(dev_w, layers[i] * layers[i + 1], "W fresh");
			GPU_fill_rand(dev_b, layers[i + 1], 0.1f); // zero initilizaton is fine for biases
			// push into vector
			w.push_back(dev_w);
			b.push_back(dev_b);
			// intermediate results arrays
			hipMalloc((void**)&dev_g, (layers[i + 1]) * sizeof(double));
			checkCUDAErrorWithLine("Cuda malloc failed!");
			hipMalloc((void**)&dev_a, (layers[i + 1]) * sizeof(double));
			checkCUDAErrorWithLine("Cuda malloc failed!");
			g.push_back(dev_g);
			a.push_back(dev_a);
			// grad arrays
			hipMalloc((void**)&dev_w, (layers[i] * layers[i + 1]) * sizeof(double));
			checkCUDAErrorWithLine("Cuda malloc failed!");
			dL_dw.push_back(dev_w);
			hipMalloc((void**)&dev_b, (layers[i + 1]) * sizeof(double));
			checkCUDAErrorWithLine("Cuda malloc failed!");
			dL_db.push_back(dev_b);
			hipMalloc((void**)&dev_jac, (layers[i + 1]) * (layers[i + 1]) * sizeof(double));
			checkCUDAErrorWithLine("Cuda malloc failed!");
			da_dg.push_back(dev_jac); // da/dg has dimensions output(g) * output(g) <Jacobian>
		}
		// initilizaton cublas handle
		hipblasCreate(&handle);
		// init final grad buffer
		hipMalloc((void**)&dL_dyhat, layers[layer_count] * sizeof(double));
	}

	// C(m,n) = A(m,k) * B(k,n)
	void Net::gpu_blas_mmul(const double *A, const double *B, double *C, const int m, const int k, const int n) {
		int lda = m, ldb = k, ldc = m;
		const double alf = 1;
		const double bet = 0;
		const double *alpha = &alf;
		const double *beta = &bet;
		// Do the actual multiplication
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	}

	double* Net::forward(double *data, int n) {
		double *res = new double[layer_sizes[layer_count - 1]]();
		assert(n == input_size);
		hipMemcpy(dev_data, data, n * sizeof(double), hipMemcpyHostToDevice);
		for (int i = 0; i < layer_count; i++) {
			blocks = ceil((layer_sizes[i] + block_size - 1) / block_size);
			// clear g, a for this layer
			hipMemset(g[i], 0, layer_sizes[i] * sizeof(double));
			checkCUDAErrorWithLine("Cuda memset failed!");
			hipMemset(a[i], 0, layer_sizes[i] * sizeof(double));
			checkCUDAErrorWithLine("Cuda memset failed!");
			int block_size_tmp = 16;
			int col_blocks = (1 + block_size_tmp - 1) / block_size_tmp, row_col = (layer_sizes[i]+ block_size_tmp - 1)/ block_size_tmp;
			// matrix multiplication
			if (!i) { // first iteration, so a[i] hasn't been set yet
				gpu_blas_mmul(dev_data, w[i], g[i], 1, input_size, layer_sizes[i]);
				checkCUDAErrorWithLine("gpu mult failed!");
			}
			else {
				gpu_blas_mmul(a[i - 1], w[i], g[i], 1, layer_sizes[i-1], layer_sizes[i]);
				checkCUDAErrorWithLine("gpu mult failed!");
			}
			// bias addition
			bias_addition << <blocks, block_size >> > (layer_sizes[i], g[i], b[i], g[i]); // put result back into g[i]
			checkCUDAErrorWithLine("bias addition failed!");
			if (i != layer_count - 1) {
				// relu activation
				relu_activation << <blocks, block_size >> > (layer_sizes[i], g[i], a[i]);
				checkCUDAErrorWithLine("relu failed!");
			}
			else {
				exp_copy << <blocks, block_size >> > (layer_sizes[i], a[i], g[i]);
				checkCUDAErrorWithLine("exp copy failed!");
				// todo move this to the gpu this later
				double *tmp = new double[layer_sizes[i]];
				double exp_sum = 0;
				hipMemcpy(tmp, a[i], layer_sizes[i] * sizeof(double), hipMemcpyDeviceToHost);
				for (int pos = 0; pos < layer_sizes[i]; pos++)
					exp_sum += tmp[pos];
				delete[] tmp;
				// modified scan to get the exponential sum of all elements (P1 of assignment used!!)
				// softmax activation
				checkCUDAErrorWithLine("Cuda memcpy failed!");
				softmax_activation << <blocks, block_size >> > (layer_sizes[i], g[i], a[i], exp_sum);
				checkCUDAErrorWithLine("softmax failed!");
			}
		}
		hipMemcpy(res, a[layer_count - 1], layer_sizes[layer_count - 1] * sizeof(double), hipMemcpyDeviceToHost);
		checkCUDAErrorWithLine("Cuda res memcpy failed!");
		return res;
	}

	void Net::backprop(double *y) {
		// calculate loss grad
		blocks = ceil((layer_sizes[layer_count - 1] + block_size - 1) / block_size);
		bias_addition <<<blocks, block_size >> > (layer_sizes[layer_count - 1], a[layer_count - 1], y, dL_dyhat, -1);
		// call softmax with correct number of threads
		for (int i = layer_count - 1; i >= 0; i--) {
			blocks = ceil((layer_sizes[i] + block_size - 1) / block_size);
			if (i == layer_count - 1) { // softmax grad
				dim3 gridDim(blocks, blocks);
				dim3 blockDim(block_size, block_size);
				softmax_grad <<< gridDim, blockDim >>> (layer_sizes[i], g[i], da_dg[i]);
			}
			else { // relu grad
				relu_grad <<< blocks, block_size >>> (layer_sizes[i], g[i], da_dg[i]);
			}
			// 
		}
	}

	double Net::loss(int *y) {
		return -1;
	}
	Net::~Net() {
		// free weights and biases
		for (auto x : w)
			hipFree(x);
		for (auto x : b)
			hipFree(x);
		// intermediate values
		for (auto x : g)
			hipFree(x);
		for (auto x : a)
			hipFree(x);
		// grads
		for (auto x : dL_dw) 
			hipFree(x);
		for (auto x : dL_db)
			hipFree(x);
		for (auto x : da_dg)
			hipFree(x);
		hipFree(dL_dyhat);
		hipFree(dev_data);
		// clean culbas hand
		hipblasDestroy(handle);
	}
}
